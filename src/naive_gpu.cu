#include "hip/hip_runtime.h"
#include <shapeDescriptor/shapeDescriptor.h>
#include <iostream>
#include <random>

#ifdef DESCRIPTOR_CUDA_KERNELS_ENABLED
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#endif

// Threads per block in each dimension
#define BLOCKX 1

// Sample rates
// #define N_POINTCLOUD_SAMPLES 5000000
// #define N_KEYPOINTS 500
#define N_POINTCLOUD_SAMPLES 5000
#define N_KEYPOINTS 10

// Randomness
#define RANDOM_SEED 22

// Host variables
const uint32_t h_n_pointcloud_samples = N_POINTCLOUD_SAMPLES;

// Device variables
__constant__ int d_n_pointcloud_samples;  // Could just leave this as 

// Define CUDA grid and block dimensions
uint32_t dim_block = BLOCKX;
uint32_t dim_grid = 1;

__global__ void calculate_descriptors(ShapeDescriptor::gpu::PointCloud device_point_cloud, ShapeDescriptor::gpu::array<uint32_t> device_keypoint_indexes, ShapeDescriptor::gpu::array<uint32_t> device_output_array) {
    uint32_t keypoints_per_thread = N_KEYPOINTS / blockDim.x;

    for (uint32_t i = threadIdx.x * keypoints_per_thread; i < (threadIdx.x + 1) * keypoints_per_thread; i++)
    {
        if (i >= N_KEYPOINTS) break;

        float3 keypoint_vertex = device_point_cloud.vertices.at(device_keypoint_indexes[i]);
        float3 keypoint_normal = device_point_cloud.normals.at(device_keypoint_indexes[i]);
        
        // ShapeDescriptor::OrientedPoint keypoint;
        // keypoint.vertex = keypoint_vertex;
        // keypoint.normal = keypoint_normal;

        device_output_array.content[i] = keypoint_vertex.x;
    }
}

int main(int argc, char **argv) {
    if (argc == 1)
    {
        std::cout << "Usage: simple_gpu [file_to_read.obj/.ply/.off]" << std::endl;
        return 1;
    }

    // Load mesh
    std::string fileToRead = std::string(argv[1]);
    ShapeDescriptor::cpu::Mesh h_mesh = ShapeDescriptor::loadMesh(fileToRead, ShapeDescriptor::RecomputeNormals::RECOMPUTE_IF_MISSING);

    // // Copy mesh to GPU
    // ShapeDescriptor::gpu::Mesh d_mesh = ShapeDescriptor::copyToGPU(h_mesh);

    // Sample point cloud
    ShapeDescriptor::cpu::PointCloud h_sampled_point_cloud = ShapeDescriptor::sampleMesh(h_mesh, N_POINTCLOUD_SAMPLES, RANDOM_SEED);
    ShapeDescriptor::gpu::PointCloud d_sampled_point_cloud = ShapeDescriptor::copyToGPU(h_sampled_point_cloud);

    // Choose random subset of N_KEYPOINTS point indices from point cloud
    std::mt19937_64 random_engine(RANDOM_SEED);
    std::uniform_int_distribution<uint32_t> distribution(0, N_POINTCLOUD_SAMPLES - 1);

    ShapeDescriptor::cpu::array<uint32_t> h_keypoint_indices(N_KEYPOINTS);
    for (uint32_t i = 0; i < N_KEYPOINTS; i++) {
        h_keypoint_indices[i] = distribution(random_engine);
    }
    ShapeDescriptor::gpu::array<uint32_t> d_keypoint_indices = ShapeDescriptor::copyToGPU(h_keypoint_indices);

    // Set up device memory for output array
    ShapeDescriptor::gpu::array<uint32_t> d_output_array(N_KEYPOINTS);

    // Launch kernel
    calculate_descriptors<<<dim_grid, dim_block>>>(d_sampled_point_cloud, d_keypoint_indices, d_output_array);

    // Synchronize and check if any errors occurred
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Got CUDA error: %s\n", hipGetErrorString(err));
    }
    std::cout << "Kernel finished" << std::endl;

    // Move output to CPU
    ShapeDescriptor::cpu::array<uint32_t> h_output_array = ShapeDescriptor::copyToCPU(d_output_array);
    
    // Print output
    for (uint32_t i = 0; i < N_KEYPOINTS; i++)
    {
        std::cout << h_output_array.content[i] << std::endl;
    }

    // Does it match?
    for (uint32_t i = 0; i < N_KEYPOINTS; i++)
    {
        std::cout << h_sampled_point_cloud.vertices[h_keypoint_indices[i]].x << std::endl;
    }
    

    // Free the memory
    ShapeDescriptor::free(h_mesh);
    ShapeDescriptor::free(h_sampled_point_cloud);
    ShapeDescriptor::free(d_sampled_point_cloud);
    ShapeDescriptor::free(h_keypoint_indices);
    ShapeDescriptor::free(d_keypoint_indices);
    ShapeDescriptor::free(d_output_array);
    // ShapeDescriptor::free(d_mesh);

    return 0;
}