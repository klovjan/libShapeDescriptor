#include "hip/hip_runtime.h"
#include <shapeDescriptor/shapeDescriptor.h>

#ifdef DESCRIPTOR_CUDA_KERNELS_ENABLED
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#endif

int foo = 1;

void checkCuSolverStatus(hipsolverStatus_t status) {
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        fprintf(stderr, "Failed to execute cuSOLVER function!\n");
        exit(EXIT_FAILURE);
    }
}

__device__ inline void swap(float &a, float &b)
{
    float temp = a;
    a = b;
    b = temp;
}

__device__ inline void swapColumns(float *eigenvectors, int columnA, int columnB)
{
    for (int r = 0; r < 3; ++r)
    {
        int i1 = r + columnA * 3;
        int i2 = r + columnB * 3;
        float temp = eigenvectors[i1];
        eigenvectors[i1] = eigenvectors[i2];
        eigenvectors[i2] = temp;
    }
}

__global__ void sortEigenvectors(float *d_allEigenvectors, float *d_allEigenvalues, int batchSize)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batchSize)
        return;

    float* eigenvectors = d_allEigenvectors + idx * 3 * 3; // pointer to start of idx-th 3x3 matrix
    float* eigenvalues = d_allEigenvalues + idx * 3;       // eigenvalues for idx-th 3x3 matrix

    int i0 = 0, i1 = 1, i2 = 2;
    if (eigenvalues[i0] < eigenvalues[i1])
    {
        swap(eigenvalues[i0], eigenvalues[i1]);
        swapColumns(eigenvectors, i0, i1);
    }
    if (eigenvalues[i0] < eigenvalues[i2])
    {
        swap(eigenvalues[i0], eigenvalues[i2]);
        swapColumns(eigenvectors, i0, i2);
    }
    if (eigenvalues[i1] < eigenvalues[i2])
    {
        swap(eigenvalues[i1], eigenvalues[i2]);
        swapColumns(eigenvectors, i1, i2);
    }
}

namespace ShapeDescriptor {
namespace gpu {
    ShapeDescriptor::gpu::array<float> computeEigenVectorsMultiple(ShapeDescriptor::gpu::array<float> d_columnMajorMatrices, uint32_t nMatrices) {
        hipsolverHandle_t cusolverHandle;
        checkCuSolverStatus(hipsolverDnCreate(&cusolverHandle));

        const int n = 3; // Matrix dimension (how many columns)
        const int lda = 3; // Leading dimension (how many rows)

        // Allocate device memory for eigenvalues and workspace
        float *d_eigenvalues;
        checkCudaErrors(hipMalloc(&d_eigenvalues, nMatrices * n * sizeof(float)));
        
        // 1. Query workspace size
        int lwork = 0;

        hipsolverSyevjInfo_t params = NULL;
        checkCuSolverStatus(hipsolverDnCreateSyevjInfo(&params));

        // Perform the actual query
        checkCuSolverStatus(hipsolverDnSsyevjBatched_bufferSize(
            cusolverHandle,
            HIPSOLVER_EIG_MODE_VECTOR,
            HIPBLAS_FILL_MODE_LOWER,
            n,
            d_columnMajorMatrices.content,
            lda,
            d_eigenvalues,
            &lwork,
            params,
            nMatrices));

        float *d_work;
        checkCudaErrors(hipMalloc(&d_work, lwork * sizeof(float)));
        
        int *d_info;
        checkCudaErrors(hipMalloc(&d_info, sizeof(int)));

        // 2. Solve eigenvalue problem for each matrix
        // Compute all eigendecompositions in a single batched call (in-place)
        checkCuSolverStatus(hipsolverDnSsyevjBatched(
            cusolverHandle,
            HIPSOLVER_EIG_MODE_VECTOR,
            HIPBLAS_FILL_MODE_LOWER,
            n,
            d_columnMajorMatrices.content,
            lda,
            d_eigenvalues,
            d_work,
            lwork,
            d_info,
            params,
            nMatrices));

        // 3. Order the eigenvectors by their lengths (i.e. by their eigenvalues)
        int block = 128;
        int grid = (nMatrices + block - 1) / block;
        sortEigenvectors<<<grid, block>>>(d_columnMajorMatrices.content, d_eigenvalues, nMatrices);

        // Clean up
        checkCudaErrors(hipFree(d_eigenvalues));
        checkCudaErrors(hipFree(d_work));
        checkCudaErrors(hipFree(d_info));
        checkCuSolverStatus(hipsolverDnDestroy(cusolverHandle));

        // 4. Create output gpu::array
        return d_columnMajorMatrices;
    }
}
}